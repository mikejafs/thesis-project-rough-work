
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloWorld() {
    printf("Hello, World from GPU!\n");
}

int main() {
    // Launch kernel
    helloWorld<<<1, 1>>>();
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    return 0;
}


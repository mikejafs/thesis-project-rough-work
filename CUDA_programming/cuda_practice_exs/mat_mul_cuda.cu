
// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>


using std::cout;
using std::generate;
using std::vector;

__global__ void mat_mul(const int* a, const int *b, int *c, int N){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++){
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N){

    for (int i = 0; i < N; i++){
        for (int j = 0; j < N; j++){
            int temp = 0;
            for (int k = 0; k < N; k++){
                temp += a[i*N + k] * b[k*N + j];
            }

            assert(temp == c[i * N + j]);
        }
    }
}

int main(){

    int N = 1 << 10;  //bit shift operater (1<<n is equiv to 2^n)
    size_t bytes = N * N * sizeof(int);

    vector<int> h_a(N * N);
    vector<int> h_b(N * N);
    vector<int> h_c(N * N);

    //Initialize matrices
    generate(h_a.begin(), h_a.end(), []() {return rand() % 100;});
    generate(h_b.begin(), h_b.end(), []() {return rand() % 100;});

    //Allocate device memory
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //Copy data from host to device
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = 32;
    int BLOCKS = N / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    mat_mul<<<blocks, threads>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    verify_result(h_a, h_b, h_c, N);

    cout << "COMPLETED SUCCESSFULLY \n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
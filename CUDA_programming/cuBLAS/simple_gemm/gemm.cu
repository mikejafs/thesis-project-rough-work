//nvcc -Xcompiler -fPIC -shared -o libgemm.so gemm.cu -lcublas


#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{
    void matmul_gemm(
        float *A,
        float *B,
        float *C,
        int m, int n, int k
    ){
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // cublasStatus_t status;

        // Create the cuBLAS handle
        // status = cublasCreate(&handle);
        // if (status != CUBLAS_STATUS_SUCCESS) {
        //     return 1;
        // }


        const float alpha = 1.0f;
        const float beta = 0.0f;

        hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, m, k,
            &alpha,
            B, n,
            A, k,
            &beta,
            C, n
        );

        hipblasDestroy(handle);

        // return (status == CUBLAS_STATUS_SUCCESS) ? 0 : 2;
    }
} 
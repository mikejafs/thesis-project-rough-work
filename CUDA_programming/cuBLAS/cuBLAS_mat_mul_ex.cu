#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <math.h>

void verify_solution(float *A, float *B, float *C, int n) {
    float temp;
    float epsilon = 0.001;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            temp = 0;
            for (int k = 0; k < n; k++) {
                temp += A[k * n + i] * B[j * n + k];
            }

            // #fab means floating point absolute value
            // Check if the computed value is within an acceptable range of the expected value
            assert(fabs(C[j * n + i] - temp) < epsilon);
        }
    }
}

int main(){
    int n = 1 << 10; // 1024
    size_t bytes = n * n * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // Allocate memory
    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_c = (float *)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //Pseudo random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    // Fill the matrices with random values on the device
    hiprandGenerateUniform(prng, d_a, n * n);
    hiprandGenerateUniform(prng, d_b, n * n);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Scaling factors
    float alpha = 1.0f;
    float beta = 0.0f;

    // Calculate: c = (alpha*a) * b + (beta * c)
    // (m X n) * (n X k) = (m X k)
    // Signature: handle, operation, operation, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, 
                d_a, n, d_b, n, &beta, d_c, n);

    // Copy the result back to the host
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Verify the solution
    verify_solution(h_a, h_b, h_c, n);  

    printf("Matrix multiplication completed successfully.\n");

    return 0;
}



// block_diag_gemm.cu
// nvcc -std=c++14 -lcublasLt -shared -fPIC -o libblockdiaggemm.so block_diag_gemm.cu

#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <vector>
#include <stdexcept>

// Helper to throw on error
inline void _chk(hipblasStatus_t s, const char* m) {
    if(s != HIPBLAS_STATUS_SUCCESS) throw std::runtime_error(m);
}
inline void _chk(hipError_t e,    const char* m) {
    if(e != hipSuccess)        throw std::runtime_error(m);
}

extern "C" void block_diag_gemm_cublaslt(
    const void* const* A_ptrs,  // [batchCount]: ptr to each M_i block (m_i×m_i, col‑major)
    const int*          starts,  // [batchCount]: inclusive row start of each block in [0…N)
    const int*          stops,   // [batchCount]: exclusive row end
    const void*         X,       // full X: N×P, col‑major
    void*               Y,       // full Y: N×P, col‑major
    int                 N,       // X,Y rows
    int                 P,       // X,Y cols
    float               alpha,
    float               beta,
    int                 batchCount)
{
    // 1) create cuBLASLt handle
    hipblasLtHandle_t lt;
    _chk(hipblasLtCreate(&lt), "hipblasLtCreate failed");

    // 2) descriptors & layouts vectors
    std::vector<hipblasLtMatmulDesc_t>   descs(batchCount);
    std::vector<hipblasLtMatrixLayout_t> layoutsA(batchCount),
                                        layoutsB(batchCount),
                                        layoutsC(batchCount);

    for(int i = 0; i < batchCount; ++i) {
        int m = stops[i] - starts[i];

        // 2.1) descriptor: FP32 compute, FP32 data
        _chk(hipblasLtMatmulDescCreate(&descs[i],
               HIPBLAS_COMPUTE_32F, HIP_R_32F),
             "MatmulDescCreate");  // :contentReference[oaicite:0]{index=0}

        // 2.2) no-transpose attributes
        hipblasOperation_t opN = HIPBLAS_OP_N;
        _chk(hipblasLtMatmulDescSetAttribute(
               descs[i],
               HIPBLASLT_MATMUL_DESC_TRANSA,
               &opN, sizeof(opN)),
             "SetAttr TRANSA");
        _chk(hipblasLtMatmulDescSetAttribute(
               descs[i],
               HIPBLASLT_MATMUL_DESC_TRANSB,
               &opN, sizeof(opN)),
             "SetAttr TRANSB");

        // 2.3) layouts: A_i is m×m (lda=m), B_i is m×P (ldb=N), C_i is m×P (ldc=N)
        _chk(hipblasLtMatrixLayoutCreate(
               &layoutsA[i], HIP_R_32F, m,   m, m),
             "Layout A");
        _chk(hipblasLtMatrixLayoutCreate(
               &layoutsB[i], HIP_R_32F, m,   P, N),
             "Layout B");
        _chk(hipblasLtMatrixLayoutCreate(
               &layoutsC[i], HIP_R_32F, m,   P, N),
             "Layout C");
    }

    // 3) create preference and set max workspace bytes
    hipblasLtMatmulPreference_t pref;
    _chk(hipblasLtMatmulPreferenceCreate(&pref),
         "PrefCreate");  
    // first, figure out needed workspace per-block
    size_t maxWorkspace = 0;
    {
      hipblasLtMatmulHeuristicResult_t h;
      int                             cnt;
      for(int i = 0; i < batchCount; ++i) {
        _chk(hipblasLtMatmulAlgoGetHeuristic(
               lt,
               descs[i],
               layoutsA[i],
               layoutsB[i],
               layoutsC[i],
               layoutsC[i],
               pref,
               /*requested=*/1,
               &h,
               &cnt),
             "AlgoGetHeuristic");  // :contentReference[oaicite:1]{index=1}
        maxWorkspace = std::max(maxWorkspace, h.workspaceSize);
      }
    }
    // then set preference to allow up to that much workspace:
    _chk(hipblasLtMatmulPreferenceSetAttribute(
           pref,
           HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
           &maxWorkspace, sizeof(maxWorkspace)),
         "PrefSetAttr");  // :contentReference[oaicite:2]{index=2}

    // 4) allocate one workspace buffer, aligned by cudaMalloc
    void* workspace = nullptr;
    _chk(hipMalloc(&workspace, maxWorkspace),
         "cudaMalloc(workspace)");

    // 5) for each block: pick algo + launch matmul
    for(int i = 0; i < batchCount; ++i) {
        int m = stops[i] - starts[i];
        // slice pointers into X,Y (col‑major, so just advance by starts[i] rows)
        const void* Bs = (const char*)X + starts[i] * sizeof(float);
        void*       Cs = (      char*)Y + starts[i] * sizeof(float);

        // 5.1) heuristic again (now with full pref)
        hipblasLtMatmulHeuristicResult_t heuristic;
        int                             cnt;
        _chk(hipblasLtMatmulAlgoGetHeuristic(
               lt,
               descs[i],
               layoutsA[i],
               layoutsB[i],
               layoutsC[i],
               layoutsC[i],
               pref,
               1, &heuristic, &cnt),
             "AlgoGetHeuristic(2)");
        if(cnt == 0) throw std::runtime_error("no algo found");

        // 5.2) actual matmul launch
        _chk(hipblasLtMatmul(
               lt,
               descs[i],
               &alpha,
               A_ptrs[i], layoutsA[i],
               Bs,         layoutsB[i],
               &beta,
               Cs,         layoutsC[i],
               Cs,         layoutsC[i],
               &heuristic.algo,
               &heuristic, 1,
               0 /*stream*/  // use default stream
             ), "hipblasLtMatmul");  // :contentReference[oaicite:3]{index=3}
    }

    // 6) clean up
    hipFree(workspace);
    for(int i = 0; i < batchCount; ++i) {
        hipblasLtMatmulDescDestroy(   descs[i]);
        hipblasLtMatrixLayoutDestroy(layoutsA[i]);
        hipblasLtMatrixLayoutDestroy(layoutsB[i]);
        hipblasLtMatrixLayoutDestroy(layoutsC[i]);
    }
    hipblasLtMatmulPreferenceDestroy(pref);
    hipblasLtDestroy(lt);
}
